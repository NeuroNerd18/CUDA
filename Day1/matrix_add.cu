#include <iostream>
#include <hip/hip_runtime.h>


__global__ void matrix_add(const float *A , const float *B ,  float *C , int rows , int cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(row < rows && col < cols){
        int i = row * cols + col;
        C[i] = A[i] + B[i];
    }

}

int main() {
    const int rows = 2;
    const int cols = 3;
    const int size = rows * cols * sizeof(float);

    float A[rows * cols] = {1,2,3,4,5,6};
    float B[rows * cols] = {1,2,3,4,5,6};
    float C[rows * cols];

    float *d_a , *d_b , *d_c;
    hipMalloc(&d_a , size);
    hipMalloc(&d_b , size);
    hipMalloc(&d_c , size);

    hipMemcpy(d_a , A, size , hipMemcpyHostToDevice);
    hipMemcpy(d_b , B, size , hipMemcpyHostToDevice);
    
    dim3 threadsperblock(16,16);
    dim3 blockspergrid((cols + 15)/16 , (rows +15)/16);

    matrix_add<<<blockspergrid , threadsperblock>>>(d_a , d_b , d_c, rows , cols);

    hipMemcpy(C , d_c , size , hipMemcpyDeviceToHost);

    for(int i = 0 ; i < rows ; i++){
        for(int j = 0 ; j < cols ; j++){
            std::cout<<C[i*cols + j]<<std::endl;
        }
        std::cout<<std::endl;
    } 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
